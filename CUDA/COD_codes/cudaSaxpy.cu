// https://student.cs.uwaterloo.ca/~cs450/w18/public/cudaSaxpy.pdf
/*
TODO more general https://developer.nvidia.com/blog/six-ways-saxpy/
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <math.h>
// #include <stdlib.h>
/*
differences __device__,etc. https://stackoverflow.com/a/39448797/21294350
*/
__global__
void saxpy(int n, float a, float *x, float *y)
{
	// http://users.wfu.edu/choss/CUDA/docs/Lecture%205.pdf p8 blockDim here is 256
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}
__host__ int main(void)
{
	// not too large

	// long long int N = 2147483647l+1;
	// long long int N = 2147483647l/(1<<3);
	// long long int N = 2147483647-255; // out of memory

	/*
	1. some value may be placed on cache and registers because this exceeds 
	"Total amount of global memory:                 3725 MBytes (3906076672 bytes)". in 
	`/opt/cuda/extras/demo_suite/deviceQuery` from https://linuxconfig.org/how-to-get-cuda-cores-count-on-linux
	2. check by `/opt/cuda/extras/compute-sanitizer/compute-sanitizer --tool memcheck cudaSaxpy.o`
	not use old CUDA-MEMCHECK https://stackoverflow.com/a/75973968/21294350
	*/
	// long long int N = 488259584l/2+1;
	long long int N = 488259584l/4;
	// long long int N = 488259584l;
	printf("%lld\n%d\n",N,2147483647/(1<<3));
	// long long int N = 256*256;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	// Perform SAXPY on 1M elements
	saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
	float maxError = 0.0f;
	for (int i = 0; i < N; i++){
		/*
		1. here can't recognize `max` in stdlib in clang https://stackoverflow.com/a/4234022/21294350
		2. directly inline max defined in https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INT.html
		3. similar to what this https://stackoverflow.com/a/54144675/21294350 says, 
		Candidate function not viable: call to __device__ function from __host__ function
		*/
		maxError = fmaxf(maxError, abs(y[i]-4.0f));
		// maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}
