#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cuda/semaphore>
#include <cuda/atomic>

__device__ uint32_t something_very_slow(uint32_t x) {
  for (uint32_t i = 0; i / 1e7 < 1; ++i) {
    x *= 13;
    x += 1;
    x %= 123456789;
  }
  return x;
}

__device__ cuda::binary_semaphore<cuda::thread_scope_block> lock{1};
__device__ cuda::atomic<uint32_t, cuda::thread_scope_block> mask{0};
__device__ cuda::atomic<uint32_t, cuda::thread_scope_block> clobber{0};

__global__ void starvation_libcudacxx() {
  lock.acquire();
  printf("start thread %d\n", threadIdx.x);
  bool cont = false;
  do {
    printf("step thread %d\n", threadIdx.x);
    lock.release();
    clobber.fetch_add(something_very_slow(clobber.load()) + threadIdx.x);
    cont = mask.fetch_add(threadIdx.x) == 0;
    lock.acquire();
  } while (cont);
  printf("done: %d\n", clobber.load());
  lock.release();
}

__global__ void starvation_legacy() {
  __shared__ uint32_t lock, mask, clobber;
  if (threadIdx.x == 0) {
    lock = mask = clobber = 0;
  }
  __syncthreads();
  while (atomicCAS(&lock, 0, 1) == 1) {
  }
  printf("start thread %d\n", threadIdx.x);
  bool cont = false;
  do {
    printf("step thread %d\n", threadIdx.x);
    atomicExch(&lock, 0);
    atomicAdd(&clobber, something_very_slow(atomicAdd(&clobber, 0)) + threadIdx.x);
    cont = atomicAdd(&mask, threadIdx.x) == 0;
    while (atomicCAS(&lock, 0, 1) == 1) {
    }
  } while (cont);
  printf("done: %d\n", atomicAdd(&clobber, 0));
  atomicExch(&lock, 0);
}

int main() {
  starvation_libcudacxx<<<1, 2>>>();
  starvation_legacy<<<1, 2>>>();
  hipDeviceSynchronize();
}
