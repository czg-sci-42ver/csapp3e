#include "hip/hip_runtime.h"
// https://stackoverflow.com/q/76497234/21294350
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cuda/semaphore>
#include <cuda/atomic>

__device__ uint32_t something_very_slow(uint32_t x) {
  for (uint32_t i = 0; i / 1e7 < 1; ++i) {
    x *= 13;
    x += 1;
    x %= 123456789;
  }
  return x;
}

__device__ cuda::binary_semaphore<cuda::thread_scope_block> lock{1};
__device__ cuda::atomic<uint32_t, cuda::thread_scope_block> mask{0};
__device__ cuda::atomic<uint32_t, cuda::thread_scope_block> clobber{0};

__global__ void starvation_libcudacxx() {
  lock.acquire();
  printf("start thread %d\n", threadIdx.x);
  bool cont = false;
  printf("step thread %d\n", threadIdx.x);
  lock.release();
  printf("release lock\n");
  clobber.fetch_add(something_very_slow(clobber.load()) + threadIdx.x,cuda::memory_order_seq_cst);
  cont = mask.fetch_add(threadIdx.x) == 0;
  printf("done: %d\n", clobber.load());
}

__global__ void starvation_legacy() {
  __shared__ uint32_t lock, mask, clobber;
  if (threadIdx.x == 0) {
    lock = mask = clobber = 0;
  }
  __syncthreads();
  while (atomicCAS(&lock, 0, 1) == 1) {
  }
  printf("starvation_legacy start thread %d\n", threadIdx.x);
  bool cont = false;
  do {
    printf("starvation_legacy step thread %d\n", threadIdx.x);
    atomicExch(&lock, 0);
    atomicAdd(&clobber, something_very_slow(atomicAdd(&clobber, 0)) + threadIdx.x);
    cont = atomicAdd(&mask, threadIdx.x) == 0;
    while (atomicCAS(&lock, 0, 1) == 1) {
    }
  } while (cont);
  printf("done: %d\n", atomicAdd(&clobber, 0));
  atomicExch(&lock, 0);
}

int main() {
  while (1) {
    starvation_libcudacxx<<<1, 2>>>();
  }
}
