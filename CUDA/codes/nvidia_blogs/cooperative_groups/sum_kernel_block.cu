#include "hip/hip_runtime.h"
// https://developer.nvidia.com/blog/cooperative-groups/

/*
other references:
ref_1: https://developer.nvidia.com/blog/even-easier-introduction-cuda/
*/
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
using namespace cooperative_groups;
__device__ int reduce_sum(thread_group g, int *temp, int val) {
  int lane = g.thread_rank();

  // Each iteration halves the number of active threads
  // Each thread adds its partial sum[i] to sum[lane+i]
  for (int i = g.size() / 2; i > 0; i /= 2) {
    temp[lane] = val;
    g.sync(); // wait for all threads to store
    if (lane < i)
      val += temp[lane + i];
    g.sync(); // wait for all threads to load
  }
  return val; // note: only thread 0 will return full sum
}
__device__ int thread_sum(int *input, int n) {
  int sum = 0;
// printf("%d; %d\n",blockIdx.x * blockDim.x + threadIdx.x,blockDim.x *
// gridDim.x);
#if __CUDA_ARCH__ >= 200
// std::cout << blockIdx.x * blockDim.x + threadIdx.x << "; " << blockDim.x *
// gridDim.x << std::endl; printf("%d; %d\n",blockIdx.x * blockDim.x +
// threadIdx.x,blockDim.x * gridDim.x);
#endif

  /*
  `i += blockDim.x * gridDim.x` -> grid-stride loop in ref_1

  2. here maybe manually selected to make the whole sum in sum_kernel_block is n
  only calculate n/4 sums.
  */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n / 4;
       i += blockDim.x * gridDim.x) // only calculate once
  {
    // if (i>=n/4)
    // 	printf("calculate more");
    int4 in = ((int4 *)input)[i];
    sum += in.x + in.y + in.z + in.w;
  }
  return sum;
}

__global__ void sum_kernel_block(int *sum, int *input, int n) {
  int my_sum = thread_sum(input, n);
  // printf("my_sum: %d\n",my_sum);

  extern __shared__ int temp[];
  auto g = this_thread_block();
  int block_sum = reduce_sum(g, temp, my_sum);

  // only add once the current thread block sum
  if (g.thread_rank() == 0)
    atomicAdd(sum, block_sum);

  thread_group tile32 = tiled_partition(this_thread_block(), 32);
  /*
  1. here thread_group only corresponds to the thread related one by "the threads of rank 0 in each tile4 group"
  in https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=cg#tiled-partition
  */
  thread_group tile4 = tiled_partition(tile32, 4);
  if (tile4.thread_rank() == 0)
    printf("Hello from tile4 rank 0: %d\n", this_thread_block().thread_rank());
}

int main() {
  // int n = 1<<24;
  int n = 256 * (1 << 3);
  int blockSize = 256;
  int nBlocks = (n + blockSize - 1) / blockSize;
  int sharedBytes = blockSize * sizeof(int);

  int *sum, *data;
  hipMallocManaged(&sum, sizeof(int));
  hipMallocManaged(&data, n * sizeof(int));
  std::fill_n(data, n, 1); // initialize data
  hipMemset(sum, 0, sizeof(int));

  sum_kernel_block<<<nBlocks, blockSize, sharedBytes>>>(sum, data, n);
  /*
  see ../C_Programming_Guide/Broadcast

  Also see
  https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#concurrent-execution-between-host-and-device
  return control to the host thread before the device completes the requested
  task

  also see ref_1
  */
  hipDeviceSynchronize();
  printf("equal: %d, sum= %d\n", *sum == n, *sum);
  // ref_1
  hipFree(sum);
  hipFree(data);
  return 0;
}
