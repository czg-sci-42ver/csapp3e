#include "hip/hip_runtime.h"
// https://developer.nvidia.com/blog/cooperative-groups/

/*
other references:
ref_1: https://developer.nvidia.com/blog/even-easier-introduction-cuda/
*/
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
using namespace cooperative_groups;

template <int tile_sz>
__device__ int reduce_sum_tile_shfl(thread_block_tile<tile_sz> g, int val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        val += g.shfl_down(val, i);
    }

    return val; // note: only thread 0 will return full sum
}

__device__ int thread_sum(int *input, int n) {
  int sum = 0;
// printf("%d; %d\n",blockIdx.x * blockDim.x + threadIdx.x,blockDim.x *
// gridDim.x);
#if __CUDA_ARCH__ >= 200
// std::cout << blockIdx.x * blockDim.x + threadIdx.x << "; " << blockDim.x *
// gridDim.x << std::endl; printf("%d; %d\n",blockIdx.x * blockDim.x +
// threadIdx.x,blockDim.x * gridDim.x);
#endif

  /*
  `i += blockDim.x * gridDim.x` -> grid-stride loop in ref_1

  2. here maybe manually selected to make the whole sum in sum_kernel_block is n
  only calculate n/4 sums.
  */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n / 4;
       i += blockDim.x * gridDim.x) // only calculate once
  {
    // if (i>=n/4)
    // 	printf("calculate more");
    int4 in = ((int4 *)input)[i];
    sum += in.x + in.y + in.z + in.w;
  }
  return sum;
}

template<int tile_sz>
__global__ void sum_kernel_tile_shfl(int *sum, int *input, int n)
{
    int my_sum = thread_sum(input, n);

    auto tile = tiled_partition<tile_sz>(this_thread_block());
    int tile_sum = reduce_sum_tile_shfl<tile_sz>(tile, my_sum);

    if (tile.thread_rank() == 0) atomicAdd(sum, tile_sum);
}

int main() {
  // int n = 1<<24;
  int n = 256 * (1 << 3);
  int blockSize = 256;
  int nBlocks = (n + blockSize - 1) / blockSize;
  int sharedBytes = blockSize * sizeof(int);

  int *sum, *data;
  hipMallocManaged(&sum, sizeof(int));
  hipMallocManaged(&data, n * sizeof(int));
  std::fill_n(data, n, 1); // initialize data
  hipMemset(sum, 0, sizeof(int));

  sum_kernel_tile_shfl<32><<<nBlocks, blockSize, sharedBytes>>>(sum, data, n);
  /*
  see ../C_Programming_Guide/Broadcast

  Also see
  https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#concurrent-execution-between-host-and-device
  return control to the host thread before the device completes the requested
  task

  also see ref_1
  */
  hipDeviceSynchronize();
  printf("equal: %d, sum= %d\n", *sum == n, *sum);
  // ref_1
  hipFree(sum);
  hipFree(data);
  return 0;
}
