#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
using namespace cooperative_groups;
__device__ int atomicAggInc(int *ctr) {
  auto g = coalesced_threads();
  int warp_res;
  if(g.thread_rank() == 0){
    printf("old: %d\n",*ctr);
    warp_res = atomicAdd(ctr, g.size());
    printf("assign: %d %d\n",warp_res,*ctr);
  }
  return g.shfl(warp_res, 0) + g.thread_rank();
}
__global__ void filter_k(int *dst,int *nres, const int *src, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  extern __shared__ int temp[];
  if(i >= n)
    return;
  if(src[i] > 0){
    temp[i] = atomicAggInc(nres);
    dst[temp[i]] = src[i];
    printf("%dth thread nres: %d\n",i,temp[i]);
  }
}
int main() {
  int warp_size = 32;
  int n = warp_size << 3;
  int block_size = 32;
  int grid_size = (n + block_size -1)/block_size;
  int *dst,*nres;
  int *src;
  int sharedBytes = block_size * sizeof(int);

  hipMallocManaged(&dst,block_size*sizeof(int));
  hipMallocManaged(&nres,sizeof(int));
  hipMallocManaged(&src,n*sizeof(int));

  hipMemset(nres, 0, sizeof(int));
  hipMemset(src, 1, n*sizeof(int));
  hipMemset(dst, 0, block_size*sizeof(int));
  filter_k<<<grid_size,block_size,sharedBytes>>>(dst,nres,src,n);
  hipDeviceSynchronize();
}
