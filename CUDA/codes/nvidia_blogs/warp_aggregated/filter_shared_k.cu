#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
using namespace cooperative_groups;
__device__ int atomicAggInc(int *ctr) {
  auto g = coalesced_threads();
  int warp_res;
  if(g.thread_rank() == 0){
    printf("old: %d\n",*ctr);
    warp_res = atomicAdd(ctr, g.size());
    printf("assign: %d %d\n",warp_res,*ctr);
  }
  return g.shfl(warp_res, 0) + g.thread_rank();
}

#define NPER_THREAD 4
#define BS 8
__global__ 
void filter_shared_k(int *dst, int *nres, const int* src, int n) {
  __shared__ int l_n;
  /*
  TODO what does NPER_THREAD * BS do?
  */
  int i = blockIdx.x * (NPER_THREAD * BS) + threadIdx.x;

  for (int iter = 0; iter < NPER_THREAD; iter++) {
    // zero the counter
    if (threadIdx.x == 0)
      l_n = 0;
    __syncthreads();

    // get the value, evaluate the predicate, and
    // increment the counter if needed
    int d, pos;

    if(i < n) {
      d = src[i];
      if(d > 0)
        pos = atomicAdd(&l_n, 1);
    }
    __syncthreads();

    // leader increments the global counter
    if(threadIdx.x == 0)
      l_n = atomicAdd(nres, l_n);
    __syncthreads();

    // threads with true predicates write their elements
    if(i < n && d > 0) {
      pos += l_n; // increment local pos by global counter
      dst[pos] = d;
    }
    __syncthreads();

    i += BS;
  }
}
int main() {
  // int warp_size = 32;
  int grid_size = 8;
  int block_size = NPER_THREAD * BS;
  int n = grid_size*block_size;
  int *dst,*nres;
  int *src;
  int sharedBytes = block_size * sizeof(int);

  hipMallocManaged(&dst,block_size*sizeof(int));
  hipMallocManaged(&nres,sizeof(int));
  hipMallocManaged(&src,n*sizeof(int));

  hipMemset(nres, 0, sizeof(int));
  hipMemset(src, 1, n*sizeof(int));
  hipMemset(dst, 0, block_size*sizeof(int));
  filter_shared_k<<<grid_size,block_size,sharedBytes>>>(dst,nres,src,n);
  hipDeviceSynchronize();
}
