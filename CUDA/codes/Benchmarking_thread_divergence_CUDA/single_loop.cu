
#include <hip/hip_runtime.h>
#define EXPR_INNER 1.3333f
#define EXPR_OUTER 2.3333f
#define llong long long int
#define N_PREHEAT 10
#define N_UNROLL 20

__global__ void single_loop(int* limits, float* out,
		llong* timer) {


	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	int M = limits[threadIdx.x];

	float sum = out[tid];


#pragma unroll

	for (int k = 0; k < N_PREHEAT; k++)

		for (int i = 0; i < M; i++) {

			sum += EXPR_INNER;

		}


	__syncthreads();

	llong start = clock64();


#pragma unroll

	for (int k = 0; k < N_UNROLL; ++k)

		for (int i = 0; i < M; i++) {

			sum += EXPR_INNER;

		}


	llong stop = clock64();

	__syncthreads();


	out[tid] = sum;

	timer[2 * tid] = start;

	timer[2 * tid + 1] = stop;

}
int main(){
	return 0;
}
